#include "hip/hip_runtime.h"
#include <float.h>
#include <hiprand/hiprand_kernel.h>

#include "device_math.cuh"

#define LOG0 -FLT_MAX

/// a nan-safe logarithm
__device__ __host__
float safeLog( float x )
{
    if ( x <= 0 )
        return LOG0 ;
    else
        return log(x) ;
}

/// evaluate generalized logistic function
__device__ __host__ float
logistic_function(float x, float lower, float upper, float beta, float tau)
{
    float y = (upper-lower)/(1+exp(-beta*(x-tau) ) ) ;
    return y ;
}

/// product of two 2x2 matrices
__device__ void
matmultiply2(float *A, float *B, float *X){
    X[0] = A[0]*B[0] + A[2]*B[1] ;
    X[1] = A[1]*B[0] + A[3]*B[1] ;
    X[2] = A[0]*B[2] + A[2]*B[3] ;
    X[3] = A[1]*B[2] + A[3]*B[3] ;
}

/// determinant of a 2x2 matrix
__host__ __device__ float
det2(float *A){
    return A[0]*A[3] - A[2]*A[1] ;
}

/// determinant of a 3x3 matrix
__host__ __device__ float
det3(float *A){
    return (A[0]*A[4]*A[8] + A[3]*A[7]*A[2] + A[6]*A[1]*A[5])
        - (A[0]*A[7]*A[5] + A[3]*A[1]*A[8] + A[6]*A[4]*A[2]) ;
}

/// determinant of a 4x4 matrix
__host__ __device__ float
det4(float *A)
{
    float det=0;
    det+=A[0]*((A[5]*A[10]*A[15]+A[9]*A[14]*A[7]+A[13]*A[6]*A[11])-(A[5]*A[14]*A[11]-A[9]*A[6]*A[15]-A[13]*A[10]*A[7]));
    det+=A[4]*((A[1]*A[14]*A[11]+A[9]*A[2]*A[15]+A[13]*A[10]*A[3])-(A[1]*A[10]*A[15]-A[9]*A[14]*A[3]-A[13]*A[2]*A[11]));
    det+=A[8]*((A[1]*A[6]*A[15]+A[5]*A[14]*A[3]+A[13]*A[2]*A[7])-(A[1]*A[14]*A[7]-A[5]*A[2]*A[15]-A[13]*A[6]*A[3]));
    det+=A[12]*((A[1]*A[10]*A[7]+A[5]*A[2]*A[12]+A[9]*A[10]*A[3])-(A[1]*A[10]*A[12]-A[5]*A[10]*A[3]-A[9]*A[2]*A[7]));
    return det ;
}

/// invert a 2x2 matrix
__device__ __host__ void
invert_matrix2(float *A, float *A_inv)
{
    float det = det2(A) ;
    A_inv[0] = A[3]/det ;
    A_inv[1] = -A[1]/det ;
    A_inv[2] = -A[2]/det ;
    A_inv[3] = A[0]/det ;
}

/// invert a 3x3 matrix
__device__ void
invert_matrix3(float *A, float* A_inv){
    float det = det3(A) ;
    A_inv[0] = (A[4]*A[8] - A[7]*A[5])/det ;
    A_inv[1] = (A[7]*A[2] - A[1]*A[8])/det ;
    A_inv[2] = (A[1]*A[5] - A[4]*A[2])/det ;
    A_inv[3] = (A[6]*A[5] - A[3]*A[8])/det ;
    A_inv[4] = (A[0]*A[8] - A[6]*A[2])/det ;
    A_inv[5] = (A[2]*A[3] - A[0]*A[5])/det ;
    A_inv[6] = (A[3]*A[7] - A[6]*A[4])/det ;
    A_inv[7] = (A[6]*A[1] - A[0]*A[7])/det ;
    A_inv[8] = (A[0]*A[4] - A[3]*A[1])/det ;
}

/// invert a 4x4 matrix
__device__ void
invert_matrix4( float *A, float *Ainv)
{
    Ainv[0] = (A[5] * A[15] * A[10] - A[5] * A[11] * A[14] - A[7] * A[13] * A[10] + A[11] * A[6] * A[13] - A[15] * A[6] * A[9] + A[7] * A[9] * A[14]) / (A[0] * A[5] * A[15] * A[10] - A[0] * A[5] * A[11] * A[14] - A[0] * A[7] * A[13] * A[10] + A[0] * A[11] * A[6] * A[13] - A[0] * A[15] * A[6] * A[9] + A[0] * A[7] * A[9] * A[14] + A[5] * A[3] * A[8] * A[14] - A[5] * A[15] * A[2] * A[8] + A[5] * A[11] * A[2] * A[12] - A[5] * A[3] * A[12] * A[10] - A[15] * A[10] * A[1] * A[4] + A[15] * A[6] * A[1] * A[8] + A[15] * A[2] * A[4] * A[9] + A[3] * A[12] * A[6] * A[9] + A[7] * A[13] * A[2] * A[8] + A[7] * A[1] * A[12] * A[10] + A[3] * A[4] * A[13] * A[10] + A[11] * A[14] * A[1] * A[4] - A[11] * A[6] * A[1] * A[12] - A[11] * A[2] * A[4] * A[13] - A[3] * A[8] * A[6] * A[13] - A[7] * A[9] * A[2] * A[12] - A[7] * A[1] * A[8] * A[14] - A[3] * A[4] * A[9] * A[14]);
    Ainv[1] = -(A[15] * A[10] * A[1] - A[11] * A[14] * A[1] + A[3] * A[9] * A[14] - A[15] * A[2] * A[9] - A[3] * A[13] * A[10] + A[11] * A[2] * A[13]) / (A[0] * A[5] * A[15] * A[10] - A[0] * A[5] * A[11] * A[14] - A[0] * A[7] * A[13] * A[10] + A[0] * A[11] * A[6] * A[13] - A[0] * A[15] * A[6] * A[9] + A[0] * A[7] * A[9] * A[14] + A[5] * A[3] * A[8] * A[14] - A[5] * A[15] * A[2] * A[8] + A[5] * A[11] * A[2] * A[12] - A[5] * A[3] * A[12] * A[10] - A[15] * A[10] * A[1] * A[4] + A[15] * A[6] * A[1] * A[8] + A[15] * A[2] * A[4] * A[9] + A[3] * A[12] * A[6] * A[9] + A[7] * A[13] * A[2] * A[8] + A[7] * A[1] * A[12] * A[10] + A[3] * A[4] * A[13] * A[10] + A[11] * A[14] * A[1] * A[4] - A[11] * A[6] * A[1] * A[12] - A[11] * A[2] * A[4] * A[13] - A[3] * A[8] * A[6] * A[13] - A[7] * A[9] * A[2] * A[12] - A[7] * A[1] * A[8] * A[14] - A[3] * A[4] * A[9] * A[14]);
    Ainv[2] = (A[5] * A[3] * A[14] - A[5] * A[15] * A[2] + A[15] * A[6] * A[1] + A[7] * A[13] * A[2] - A[3] * A[6] * A[13] - A[7] * A[1] * A[14]) / (A[0] * A[5] * A[15] * A[10] - A[0] * A[5] * A[11] * A[14] - A[0] * A[7] * A[13] * A[10] + A[0] * A[11] * A[6] * A[13] - A[0] * A[15] * A[6] * A[9] + A[0] * A[7] * A[9] * A[14] + A[5] * A[3] * A[8] * A[14] - A[5] * A[15] * A[2] * A[8] + A[5] * A[11] * A[2] * A[12] - A[5] * A[3] * A[12] * A[10] - A[15] * A[10] * A[1] * A[4] + A[15] * A[6] * A[1] * A[8] + A[15] * A[2] * A[4] * A[9] + A[3] * A[12] * A[6] * A[9] + A[7] * A[13] * A[2] * A[8] + A[7] * A[1] * A[12] * A[10] + A[3] * A[4] * A[13] * A[10] + A[11] * A[14] * A[1] * A[4] - A[11] * A[6] * A[1] * A[12] - A[11] * A[2] * A[4] * A[13] - A[3] * A[8] * A[6] * A[13] - A[7] * A[9] * A[2] * A[12] - A[7] * A[1] * A[8] * A[14] - A[3] * A[4] * A[9] * A[14]);
    Ainv[3] = -(A[5] * A[3] * A[10] - A[5] * A[11] * A[2] - A[3] * A[6] * A[9] - A[7] * A[1] * A[10] + A[11] * A[6] * A[1] + A[7] * A[9] * A[2]) / (A[0] * A[5] * A[15] * A[10] - A[0] * A[5] * A[11] * A[14] - A[0] * A[7] * A[13] * A[10] + A[0] * A[11] * A[6] * A[13] - A[0] * A[15] * A[6] * A[9] + A[0] * A[7] * A[9] * A[14] + A[5] * A[3] * A[8] * A[14] - A[5] * A[15] * A[2] * A[8] + A[5] * A[11] * A[2] * A[12] - A[5] * A[3] * A[12] * A[10] - A[15] * A[10] * A[1] * A[4] + A[15] * A[6] * A[1] * A[8] + A[15] * A[2] * A[4] * A[9] + A[3] * A[12] * A[6] * A[9] + A[7] * A[13] * A[2] * A[8] + A[7] * A[1] * A[12] * A[10] + A[3] * A[4] * A[13] * A[10] + A[11] * A[14] * A[1] * A[4] - A[11] * A[6] * A[1] * A[12] - A[11] * A[2] * A[4] * A[13] - A[3] * A[8] * A[6] * A[13] - A[7] * A[9] * A[2] * A[12] - A[7] * A[1] * A[8] * A[14] - A[3] * A[4] * A[9] * A[14]);
    Ainv[4] = -(A[15] * A[10] * A[4] - A[15] * A[6] * A[8] - A[7] * A[12] * A[10] - A[11] * A[14] * A[4] + A[11] * A[6] * A[12] + A[7] * A[8] * A[14]) / (A[0] * A[5] * A[15] * A[10] - A[0] * A[5] * A[11] * A[14] - A[0] * A[7] * A[13] * A[10] + A[0] * A[11] * A[6] * A[13] - A[0] * A[15] * A[6] * A[9] + A[0] * A[7] * A[9] * A[14] + A[5] * A[3] * A[8] * A[14] - A[5] * A[15] * A[2] * A[8] + A[5] * A[11] * A[2] * A[12] - A[5] * A[3] * A[12] * A[10] - A[15] * A[10] * A[1] * A[4] + A[15] * A[6] * A[1] * A[8] + A[15] * A[2] * A[4] * A[9] + A[3] * A[12] * A[6] * A[9] + A[7] * A[13] * A[2] * A[8] + A[7] * A[1] * A[12] * A[10] + A[3] * A[4] * A[13] * A[10] + A[11] * A[14] * A[1] * A[4] - A[11] * A[6] * A[1] * A[12] - A[11] * A[2] * A[4] * A[13] - A[3] * A[8] * A[6] * A[13] - A[7] * A[9] * A[2] * A[12] - A[7] * A[1] * A[8] * A[14] - A[3] * A[4] * A[9] * A[14]);
    Ainv[5] = (A[0] * A[15] * A[10] - A[0] * A[11] * A[14] + A[3] * A[8] * A[14] - A[15] * A[2] * A[8] + A[11] * A[2] * A[12] - A[3] * A[12] * A[10]) / (A[0] * A[5] * A[15] * A[10] - A[0] * A[5] * A[11] * A[14] - A[0] * A[7] * A[13] * A[10] + A[0] * A[11] * A[6] * A[13] - A[0] * A[15] * A[6] * A[9] + A[0] * A[7] * A[9] * A[14] + A[5] * A[3] * A[8] * A[14] - A[5] * A[15] * A[2] * A[8] + A[5] * A[11] * A[2] * A[12] - A[5] * A[3] * A[12] * A[10] - A[15] * A[10] * A[1] * A[4] + A[15] * A[6] * A[1] * A[8] + A[15] * A[2] * A[4] * A[9] + A[3] * A[12] * A[6] * A[9] + A[7] * A[13] * A[2] * A[8] + A[7] * A[1] * A[12] * A[10] + A[3] * A[4] * A[13] * A[10] + A[11] * A[14] * A[1] * A[4] - A[11] * A[6] * A[1] * A[12] - A[11] * A[2] * A[4] * A[13] - A[3] * A[8] * A[6] * A[13] - A[7] * A[9] * A[2] * A[12] - A[7] * A[1] * A[8] * A[14] - A[3] * A[4] * A[9] * A[14]);
    Ainv[6] = -(A[0] * A[15] * A[6] - A[0] * A[7] * A[14] - A[15] * A[2] * A[4] - A[3] * A[12] * A[6] + A[3] * A[4] * A[14] + A[7] * A[2] * A[12]) / (A[0] * A[5] * A[15] * A[10] - A[0] * A[5] * A[11] * A[14] - A[0] * A[7] * A[13] * A[10] + A[0] * A[11] * A[6] * A[13] - A[0] * A[15] * A[6] * A[9] + A[0] * A[7] * A[9] * A[14] + A[5] * A[3] * A[8] * A[14] - A[5] * A[15] * A[2] * A[8] + A[5] * A[11] * A[2] * A[12] - A[5] * A[3] * A[12] * A[10] - A[15] * A[10] * A[1] * A[4] + A[15] * A[6] * A[1] * A[8] + A[15] * A[2] * A[4] * A[9] + A[3] * A[12] * A[6] * A[9] + A[7] * A[13] * A[2] * A[8] + A[7] * A[1] * A[12] * A[10] + A[3] * A[4] * A[13] * A[10] + A[11] * A[14] * A[1] * A[4] - A[11] * A[6] * A[1] * A[12] - A[11] * A[2] * A[4] * A[13] - A[3] * A[8] * A[6] * A[13] - A[7] * A[9] * A[2] * A[12] - A[7] * A[1] * A[8] * A[14] - A[3] * A[4] * A[9] * A[14]);
    Ainv[7] = (-A[0] * A[7] * A[10] + A[0] * A[11] * A[6] + A[7] * A[2] * A[8] + A[3] * A[4] * A[10] - A[11] * A[2] * A[4] - A[3] * A[8] * A[6]) / (A[0] * A[5] * A[15] * A[10] - A[0] * A[5] * A[11] * A[14] - A[0] * A[7] * A[13] * A[10] + A[0] * A[11] * A[6] * A[13] - A[0] * A[15] * A[6] * A[9] + A[0] * A[7] * A[9] * A[14] + A[5] * A[3] * A[8] * A[14] - A[5] * A[15] * A[2] * A[8] + A[5] * A[11] * A[2] * A[12] - A[5] * A[3] * A[12] * A[10] - A[15] * A[10] * A[1] * A[4] + A[15] * A[6] * A[1] * A[8] + A[15] * A[2] * A[4] * A[9] + A[3] * A[12] * A[6] * A[9] + A[7] * A[13] * A[2] * A[8] + A[7] * A[1] * A[12] * A[10] + A[3] * A[4] * A[13] * A[10] + A[11] * A[14] * A[1] * A[4] - A[11] * A[6] * A[1] * A[12] - A[11] * A[2] * A[4] * A[13] - A[3] * A[8] * A[6] * A[13] - A[7] * A[9] * A[2] * A[12] - A[7] * A[1] * A[8] * A[14] - A[3] * A[4] * A[9] * A[14]);
    Ainv[8] = (-A[5] * A[15] * A[8] + A[5] * A[11] * A[12] + A[15] * A[4] * A[9] + A[7] * A[13] * A[8] - A[11] * A[4] * A[13] - A[7] * A[9] * A[12]) / (A[0] * A[5] * A[15] * A[10] - A[0] * A[5] * A[11] * A[14] - A[0] * A[7] * A[13] * A[10] + A[0] * A[11] * A[6] * A[13] - A[0] * A[15] * A[6] * A[9] + A[0] * A[7] * A[9] * A[14] + A[5] * A[3] * A[8] * A[14] - A[5] * A[15] * A[2] * A[8] + A[5] * A[11] * A[2] * A[12] - A[5] * A[3] * A[12] * A[10] - A[15] * A[10] * A[1] * A[4] + A[15] * A[6] * A[1] * A[8] + A[15] * A[2] * A[4] * A[9] + A[3] * A[12] * A[6] * A[9] + A[7] * A[13] * A[2] * A[8] + A[7] * A[1] * A[12] * A[10] + A[3] * A[4] * A[13] * A[10] + A[11] * A[14] * A[1] * A[4] - A[11] * A[6] * A[1] * A[12] - A[11] * A[2] * A[4] * A[13] - A[3] * A[8] * A[6] * A[13] - A[7] * A[9] * A[2] * A[12] - A[7] * A[1] * A[8] * A[14] - A[3] * A[4] * A[9] * A[14]);
    Ainv[9] = -(A[0] * A[15] * A[9] - A[0] * A[11] * A[13] - A[15] * A[1] * A[8] - A[3] * A[12] * A[9] + A[11] * A[1] * A[12] + A[3] * A[8] * A[13]) / (A[0] * A[5] * A[15] * A[10] - A[0] * A[5] * A[11] * A[14] - A[0] * A[7] * A[13] * A[10] + A[0] * A[11] * A[6] * A[13] - A[0] * A[15] * A[6] * A[9] + A[0] * A[7] * A[9] * A[14] + A[5] * A[3] * A[8] * A[14] - A[5] * A[15] * A[2] * A[8] + A[5] * A[11] * A[2] * A[12] - A[5] * A[3] * A[12] * A[10] - A[15] * A[10] * A[1] * A[4] + A[15] * A[6] * A[1] * A[8] + A[15] * A[2] * A[4] * A[9] + A[3] * A[12] * A[6] * A[9] + A[7] * A[13] * A[2] * A[8] + A[7] * A[1] * A[12] * A[10] + A[3] * A[4] * A[13] * A[10] + A[11] * A[14] * A[1] * A[4] - A[11] * A[6] * A[1] * A[12] - A[11] * A[2] * A[4] * A[13] - A[3] * A[8] * A[6] * A[13] - A[7] * A[9] * A[2] * A[12] - A[7] * A[1] * A[8] * A[14] - A[3] * A[4] * A[9] * A[14]);
    Ainv[10] = (A[15] * A[0] * A[5] - A[15] * A[1] * A[4] - A[3] * A[12] * A[5] - A[7] * A[0] * A[13] + A[7] * A[1] * A[12] + A[3] * A[4] * A[13]) / (A[0] * A[5] * A[15] * A[10] - A[0] * A[5] * A[11] * A[14] - A[0] * A[7] * A[13] * A[10] + A[0] * A[11] * A[6] * A[13] - A[0] * A[15] * A[6] * A[9] + A[0] * A[7] * A[9] * A[14] + A[5] * A[3] * A[8] * A[14] - A[5] * A[15] * A[2] * A[8] + A[5] * A[11] * A[2] * A[12] - A[5] * A[3] * A[12] * A[10] - A[15] * A[10] * A[1] * A[4] + A[15] * A[6] * A[1] * A[8] + A[15] * A[2] * A[4] * A[9] + A[3] * A[12] * A[6] * A[9] + A[7] * A[13] * A[2] * A[8] + A[7] * A[1] * A[12] * A[10] + A[3] * A[4] * A[13] * A[10] + A[11] * A[14] * A[1] * A[4] - A[11] * A[6] * A[1] * A[12] - A[11] * A[2] * A[4] * A[13] - A[3] * A[8] * A[6] * A[13] - A[7] * A[9] * A[2] * A[12] - A[7] * A[1] * A[8] * A[14] - A[3] * A[4] * A[9] * A[14]);
    Ainv[11] = -(A[11] * A[0] * A[5] - A[11] * A[1] * A[4] - A[3] * A[8] * A[5] - A[7] * A[0] * A[9] + A[7] * A[1] * A[8] + A[3] * A[4] * A[9]) / (A[0] * A[5] * A[15] * A[10] - A[0] * A[5] * A[11] * A[14] - A[0] * A[7] * A[13] * A[10] + A[0] * A[11] * A[6] * A[13] - A[0] * A[15] * A[6] * A[9] + A[0] * A[7] * A[9] * A[14] + A[5] * A[3] * A[8] * A[14] - A[5] * A[15] * A[2] * A[8] + A[5] * A[11] * A[2] * A[12] - A[5] * A[3] * A[12] * A[10] - A[15] * A[10] * A[1] * A[4] + A[15] * A[6] * A[1] * A[8] + A[15] * A[2] * A[4] * A[9] + A[3] * A[12] * A[6] * A[9] + A[7] * A[13] * A[2] * A[8] + A[7] * A[1] * A[12] * A[10] + A[3] * A[4] * A[13] * A[10] + A[11] * A[14] * A[1] * A[4] - A[11] * A[6] * A[1] * A[12] - A[11] * A[2] * A[4] * A[13] - A[3] * A[8] * A[6] * A[13] - A[7] * A[9] * A[2] * A[12] - A[7] * A[1] * A[8] * A[14] - A[3] * A[4] * A[9] * A[14]);
    Ainv[12] = -(-A[5] * A[8] * A[14] + A[5] * A[12] * A[10] - A[12] * A[6] * A[9] - A[4] * A[13] * A[10] + A[8] * A[6] * A[13] + A[4] * A[9] * A[14]) / (A[0] * A[5] * A[15] * A[10] - A[0] * A[5] * A[11] * A[14] - A[0] * A[7] * A[13] * A[10] + A[0] * A[11] * A[6] * A[13] - A[0] * A[15] * A[6] * A[9] + A[0] * A[7] * A[9] * A[14] + A[5] * A[3] * A[8] * A[14] - A[5] * A[15] * A[2] * A[8] + A[5] * A[11] * A[2] * A[12] - A[5] * A[3] * A[12] * A[10] - A[15] * A[10] * A[1] * A[4] + A[15] * A[6] * A[1] * A[8] + A[15] * A[2] * A[4] * A[9] + A[3] * A[12] * A[6] * A[9] + A[7] * A[13] * A[2] * A[8] + A[7] * A[1] * A[12] * A[10] + A[3] * A[4] * A[13] * A[10] + A[11] * A[14] * A[1] * A[4] - A[11] * A[6] * A[1] * A[12] - A[11] * A[2] * A[4] * A[13] - A[3] * A[8] * A[6] * A[13] - A[7] * A[9] * A[2] * A[12] - A[7] * A[1] * A[8] * A[14] - A[3] * A[4] * A[9] * A[14]);
    Ainv[13] = (-A[0] * A[13] * A[10] + A[0] * A[9] * A[14] + A[13] * A[2] * A[8] + A[1] * A[12] * A[10] - A[9] * A[2] * A[12] - A[1] * A[8] * A[14]) / (A[0] * A[5] * A[15] * A[10] - A[0] * A[5] * A[11] * A[14] - A[0] * A[7] * A[13] * A[10] + A[0] * A[11] * A[6] * A[13] - A[0] * A[15] * A[6] * A[9] + A[0] * A[7] * A[9] * A[14] + A[5] * A[3] * A[8] * A[14] - A[5] * A[15] * A[2] * A[8] + A[5] * A[11] * A[2] * A[12] - A[5] * A[3] * A[12] * A[10] - A[15] * A[10] * A[1] * A[4] + A[15] * A[6] * A[1] * A[8] + A[15] * A[2] * A[4] * A[9] + A[3] * A[12] * A[6] * A[9] + A[7] * A[13] * A[2] * A[8] + A[7] * A[1] * A[12] * A[10] + A[3] * A[4] * A[13] * A[10] + A[11] * A[14] * A[1] * A[4] - A[11] * A[6] * A[1] * A[12] - A[11] * A[2] * A[4] * A[13] - A[3] * A[8] * A[6] * A[13] - A[7] * A[9] * A[2] * A[12] - A[7] * A[1] * A[8] * A[14] - A[3] * A[4] * A[9] * A[14]);
    Ainv[14] = -(A[14] * A[0] * A[5] - A[14] * A[1] * A[4] - A[2] * A[12] * A[5] - A[6] * A[0] * A[13] + A[6] * A[1] * A[12] + A[2] * A[4] * A[13]) / (A[0] * A[5] * A[15] * A[10] - A[0] * A[5] * A[11] * A[14] - A[0] * A[7] * A[13] * A[10] + A[0] * A[11] * A[6] * A[13] - A[0] * A[15] * A[6] * A[9] + A[0] * A[7] * A[9] * A[14] + A[5] * A[3] * A[8] * A[14] - A[5] * A[15] * A[2] * A[8] + A[5] * A[11] * A[2] * A[12] - A[5] * A[3] * A[12] * A[10] - A[15] * A[10] * A[1] * A[4] + A[15] * A[6] * A[1] * A[8] + A[15] * A[2] * A[4] * A[9] + A[3] * A[12] * A[6] * A[9] + A[7] * A[13] * A[2] * A[8] + A[7] * A[1] * A[12] * A[10] + A[3] * A[4] * A[13] * A[10] + A[11] * A[14] * A[1] * A[4] - A[11] * A[6] * A[1] * A[12] - A[11] * A[2] * A[4] * A[13] - A[3] * A[8] * A[6] * A[13] - A[7] * A[9] * A[2] * A[12] - A[7] * A[1] * A[8] * A[14] - A[3] * A[4] * A[9] * A[14]);
    Ainv[15] = 0.1e1 / (A[0] * A[5] * A[15] * A[10] - A[0] * A[5] * A[11] * A[14] - A[0] * A[7] * A[13] * A[10] + A[0] * A[11] * A[6] * A[13] - A[0] * A[15] * A[6] * A[9] + A[0] * A[7] * A[9] * A[14] + A[5] * A[3] * A[8] * A[14] - A[5] * A[15] * A[2] * A[8] + A[5] * A[11] * A[2] * A[12] - A[5] * A[3] * A[12] * A[10] - A[15] * A[10] * A[1] * A[4] + A[15] * A[6] * A[1] * A[8] + A[15] * A[2] * A[4] * A[9] + A[3] * A[12] * A[6] * A[9] + A[7] * A[13] * A[2] * A[8] + A[7] * A[1] * A[12] * A[10] + A[3] * A[4] * A[13] * A[10] + A[11] * A[14] * A[1] * A[4] - A[11] * A[6] * A[1] * A[12] - A[11] * A[2] * A[4] * A[13] - A[3] * A[8] * A[6] * A[13] - A[7] * A[9] * A[2] * A[12] - A[7] * A[1] * A[8] * A[14] - A[3] * A[4] * A[9] * A[14]) * (A[10] * A[0] * A[5] - A[10] * A[1] * A[4] - A[2] * A[8] * A[5] - A[6] * A[0] * A[9] + A[6] * A[1] * A[8] + A[2] * A[4] * A[9]);
}

/// Lower Cholesky decomposition of a square matrix.
/// No check for positive-definiteness is performed.
__device__ __host__
void cholesky(float* A, float* L, int dims){
    for ( int i = 0; i < dims*dims ; i++ )
        L[i] = 0.0 ;

    L[0] = sqrt(A[0]) ;
    for (int i = 0 ; i < dims ; i++){
        for ( int j = 0 ; j <= i ; j++){
            int ij = i + j*dims ;
            float tmp = A[ij] ;
            if ( i == j ){
                for (int k = 0 ; k < j ; k++){
                    int jk = j + k*dims ;
                    tmp -= L[jk]*L[jk] ;
                }
                L[ij] = sqrt(tmp) ;
            }
            else{
                for ( int k = 0 ; k < j ; k++){
                    int ik = i + k*dims ;
                    int jk = j + k*dims ;
                    tmp -= L[ik]*L[jk] ;
                }
                int jj = j + j*dims ;
                L[ij] = tmp/L[jj] ;
            }
        }
    }
}

__device__ __host__ void
triangular_inverse(float *L, float *Linv, int dims){
    // solve for the columns of the inverse using forward substitution
    for (int col = 0 ; col < dims ; col++ ){
        for ( int i = 0 ; i < dims ; i++ ){
            if ( i >= col ){
                float val ;
                if ( i == col )
                    val = 1 ;
                else
                    val = 0 ;

                for( int j = 0 ; j < i ; j++ )
                    val -= L[i + j*dims]*Linv[j+col*dims] ;

                Linv[i+col*dims] = val/L[i+i*dims] ;
            }
            else{
                Linv[i+col*dims] = 0.0 ;
            }
        }
    }
}

__device__ float
evalGaussian(Gaussian2D g, float2 p){
    // distance from mean
    float d[2] ;
    d[0] = g.mean[0] - p.x ;
    d[1] = g.mean[1] - p.y ;

    // inverse covariance matrix
    float S_inv[4] ;
    invert_matrix2(g.cov,S_inv);

    // determinant of covariance matrix
    float det_S = det2(g.cov) ;

    // compute exponential
    float exponent = 0.5*(d[0]*d[0]*S_inv[0]
            + d[0]*d[1]*(S_inv[1]+S_inv[2])
            + d[1]*d[1]*S_inv[3]) ;

    return exp(exponent)/sqrt(det_S)/(2*M_PI)*g.weight ;
}

__device__ float
evalLogGaussian(Gaussian2D g, float*p){
    // distance from mean
    float d[2] ;
    d[0] = g.mean[0] - p[0] ;
    d[1] = g.mean[1] - p[1] ;

    // inverse covariance matrix
    float S_inv[4] ;
    invert_matrix2(g.cov,S_inv);

    // determinant of covariance matrix
    float det_S = det2(g.cov) ;

    // compute exponential
    float exponent = 0.5*(d[0]*d[0]*S_inv[0]
            + d[0]*d[1]*(S_inv[1]+S_inv[2])
            + d[1]*d[1]*S_inv[3]) ;

    return exponent - safeLog(sqrt(det_S)) - safeLog(2*M_PI) +
                safeLog(g.weight) ;
}

template<class GaussianType>
__device__ __host__ int
getGaussianDim(GaussianType g)
{
//    int dims = sizeof(g.mean)/sizeof(float) ;
    return g.dims ;
}

template<class GaussianType>
__device__ __host__ GaussianType
sumGaussians(GaussianType a, GaussianType b)
{
    GaussianType result ;
    int dims = getGaussianDim(a) ;
    for (int i = 0 ; i < dims*dims ; i++ )
    {
        if (i < dims)
            result.mean[i] = a.mean[i] + b.mean[i] ;
        result.cov[i] = a.cov[i] + b.cov[i] ;
    }
    result.weight = a.weight + b.weight ;
    return result ;
}

template<class GaussianType>
__device__ __host__ void
clearGaussian(GaussianType &a)
{
    int dims = getGaussianDim(a) ;
    a.weight = 0 ;
    for (int i = 0 ; i < dims*dims ; i++)
    {
        if (i < dims)
            a.mean[i] = 0 ;
        a.cov[i] = 0 ;
    }
}

/// wrap an angular value to the range [-pi,pi]
__host__ __device__ float
wrapAngle(float a)
{
    float remainder = fmod(a, float(2*M_PI)) ;
    if ( remainder > M_PI )
        remainder -= 2*M_PI ;
    else if ( remainder < -M_PI )
        remainder += 2*M_PI ;
    return remainder ;
}

/// return the closest symmetric positve definite matrix for 2x2 input
__device__ void
makePositiveDefinite( float A[4] )
{
    // eigenvalues:
    float detA = A[0]*A[3] + A[1]*A[2] ;
    // check if already positive definite
    if ( detA > 0 && A[0] > 0 )
    {
        A[1] = (A[1] + A[2])/2 ;
        A[2] = A[1] ;
        return ;
    }
    float trA = A[0] + A[3] ;
    float trA2 = trA*trA ;
    float eval1 = 0.5*trA + 0.5*sqrt( trA2 - 4*detA ) ;
    float eval2 = 0.5*trA - 0.5*sqrt( trA2 - 4*detA ) ;

    // eigenvectors:
    float Q[4] ;
    if ( fabs(A[1]) > 0 )
    {
        Q[0] = eval1 - A[3] ;
        Q[1] = A[1] ;
        Q[2] = eval2 - A[3] ;
        Q[3] = A[1] ;
    }
    else if ( fabs(A[2]) > 0 )
    {
        Q[0] = A[2] ;
        Q[1] = eval1 - A[0] ;
        Q[2] = A[2] ;
        Q[3] = eval2 - A[0] ;
    }
    else
    {
        Q[0] = 1 ;
        Q[1] = 0 ;
        Q[2] = 0 ;
        Q[3] = 1 ;
    }

    // make eigenvalues positive
    if ( eval1 < 0 )
        eval1 = DBL_EPSILON ;
    if ( eval2 < 0 )
        eval2 = DBL_EPSILON ;

    // compute the approximate matrix
    A[0] = Q[0]*Q[0]*eval1 + Q[2]*Q[2]*eval2 ;
    A[1] = Q[0]*eval1*Q[1] + Q[2]*eval2*Q[3] ;
    A[2] = A[1] ;
    A[3] = Q[1]*Q[1]*eval1 + Q[3]*Q[3]*eval2 ;
}

template <int N, int N2>
__device__ float
computeMahalDist(Gaussian<N,N2> a, Gaussian<N,N2> b){
    // innovation vector
    float innov[N] ;
    for ( int i = 0 ; i < N ; i++ )
        innov[i] = a.mean[i] - b.mean[i] ;

    // innovation covariance
    float L[N2] ;
    float sigma[N2] ;
    for (int i = 0 ; i < N ; i++)
        sigma[i] = (a.cov[i]+b.cov[i])/2 ;

    // cholesky decomposition and inverse
    cholesky(sigma,L,N);
    float Linv[N2] ;
    triangular_inverse(L,Linv,N) ;

    // multiply innovation with inverse L
    // distance is sum of squares
    float dist = 0 ;
    for ( int i = 0 ; i < N ; i++ ){
        float sum = 0 ;
        for ( int j = 0 ; j <= i ; j++){
            sum += innov[j]*Linv[i+j*N] ;
        }
        dist += sum*sum ;
    }
    return dist ;
}

/// compute the Mahalanobis distance between two Gaussians
__device__ float
computeMahalDist(Gaussian2D a, Gaussian2D b)
{
    float dist = 0 ;
    float sigma_inv[4] ;
    float sigma[4] ;
    for (int i = 0 ; i <4 ; i++)
        sigma[i] = (a.cov[i] + b.cov[i])/2 ;
    invert_matrix2(sigma,sigma_inv);
    float innov[2] ;
    innov[0] = a.mean[0] - b.mean[0] ;
    innov[1] = a.mean[1] - b.mean[1] ;
    dist = innov[0]*innov[0]*sigma_inv[0] +
            innov[0]*innov[1]*(sigma_inv[1]+sigma_inv[2]) +
            innov[1]*innov[1]*sigma_inv[3] ;
    return dist ;
}

__device__ float
computeMahalDist(Gaussian3D a, Gaussian3D b)
{
    float dist = 0 ;
    float sigma_inv[9] ;
    float sigma[9] ;
    for (int i = 0 ; i <9 ; i++)
        sigma[i] = (a.cov[i] + b.cov[i])/2 ;
    invert_matrix3(sigma,sigma_inv);
    float innov[3] ;
    innov[0] = a.mean[0] - b.mean[0] ;
    innov[1] = a.mean[1] - b.mean[1] ;
    innov[2] = a.mean[1] - b.mean[1] ;
    dist = innov[0]*(sigma_inv[0]*innov[0] + sigma_inv[3]*innov[1] + sigma_inv[6]*innov[2])
            + innov[1]*(sigma_inv[1]*innov[0] + sigma_inv[4]*innov[1] + sigma_inv[7]*innov[2])
            + innov[2]*(sigma_inv[2]*innov[0] + sigma_inv[5]*innov[1] + sigma_inv[8]*innov[2]) ;
    return dist ;
}

__device__ float
computeMahalDist(Gaussian4D a, Gaussian4D b)
{
    float dist = 0 ;
    float sigma_inv[16] ;
    float sigma[16] ;
    for (int i = 0 ; i < 16 ; i++)
        sigma[i] = (a.cov[i] + b.cov[i])/2 ;
    invert_matrix4(sigma,sigma_inv) ;
    float innov[4] ;
    for ( int i = 0 ; i < 4 ; i++ )
        innov[i] = a.mean[i] - b.mean[i] ;
    dist = innov[0]*(sigma_inv[0]*innov[0] + sigma_inv[4]*innov[1] + sigma_inv[8]*innov[2] + sigma_inv[12]*innov[3])
            + innov[1]*(sigma_inv[1]*innov[0] + sigma_inv[5]*innov[1] + sigma_inv[9]*innov[2] + sigma_inv[13]*innov[3])
            + innov[2]*(sigma_inv[2]*innov[0] + sigma_inv[6]*innov[1] + sigma_inv[10]*innov[2] + sigma_inv[14]*innov[3])
            + innov[3]*(sigma_inv[3]*innov[0] + sigma_inv[7]*innov[1] + sigma_inv[11]*innov[2] + sigma_inv[15]*innov[3]) ;
    return dist ;
}

/// Compute the Hellinger distance between two Gaussians
template <class T>
__device__ float
computeHellingerDist(T a, T b)
{
    return 0 ;
}

__device__ float
computeHellingerDist( Gaussian2D a, Gaussian2D b)
{
    float dist = 0 ;
    float innov[2] ;
    float sigma[4] ;
    float detSigma ;
    float sigmaInv[4] = {1,0,0,1} ;
    innov[0] = a.mean[0] - b.mean[0] ;
    innov[1] = a.mean[1] - b.mean[1] ;
    sigma[0] = a.cov[0] + b.cov[0] ;
    sigma[1] = a.cov[1] + b.cov[1] ;
    sigma[2] = a.cov[2] + b.cov[2] ;
    sigma[3] = a.cov[3] + b.cov[3] ;
    detSigma = det2(sigma) ;
    if (detSigma > FLT_MIN)
    {
        sigmaInv[0] = sigma[3]/detSigma ;
        sigmaInv[1] = -sigma[1]/detSigma ;
        sigmaInv[2] = -sigma[2]/detSigma ;
        sigmaInv[3] = sigma[0]/detSigma ;
    }
    float epsilon = -0.25*
            (innov[0]*innov[0]*sigmaInv[0] +
             innov[0]*innov[1]*(sigmaInv[1]+sigmaInv[2]) +
             innov[1]*innov[1]*sigmaInv[3]) ;

    // determinant of half the sum of covariances
    detSigma /= 4 ;
    dist = 1/detSigma ;

    // product of covariances
    sigma[0] = a.cov[0]*b.cov[0] + a.cov[2]*b.cov[1] ;
    sigma[1] = a.cov[1]*b.cov[0] + a.cov[3]*b.cov[1] ;
    sigma[2] = a.cov[0]*b.cov[2] + a.cov[2]*b.cov[3] ;
    sigma[3] = a.cov[1]*b.cov[2] + a.cov[3]*b.cov[3] ;
    detSigma = det2(sigma) ;
    dist *= sqrt(detSigma) ;
    dist = 1 - sqrt(dist)*exp(epsilon) ;
    return dist ;
}


//__device__ void
//cholesky( float*A, float* L, int size)
//{
//    int i = size ;
//    int n_elements = 0 ;
//    while(i > 0)
//    {
//        n_elements += i ;
//        i-- ;
//    }

//    int diag_idx = 0 ;
//    int diag_inc = size ;
//    L[0] = sqrt(A[0]) ;
//    for ( i = 0 ; i < n_elements ; i++ )
//    {
//        if (i==diag_idx)
//        {
//            L[i] = A[i] ;
//            diag_idx += diag_inc ;
//            diag_inc-- ;
//        }
//    }
//}


/// device function for summations by parallel reduction in shared memory
/*!
  * Implementation based on NVIDIA whitepaper found at:
  * http://developer.download.nvidia.com/compute/cuda/1_1/Website/projects/reduction/doc/reduction.pdf
  *
  * Result is stored in sdata[0]
  \param sdata pointer to shared memory array
  \param mySum summand loaded by the thread
  \param tid thread index
  */
__device__ void
sumByReduction( volatile float* sdata, float mySum, const unsigned int tid )
{
    sdata[tid] = mySum;
    __syncthreads();

    // do reduction in shared mem
    if (tid < 128) { sdata[tid] = mySum = mySum + sdata[tid + 128]; } __syncthreads();
    if (tid <  64) { sdata[tid] = mySum = mySum + sdata[tid +  64]; } __syncthreads();

    if (tid < 32)
    {
        sdata[tid] = mySum = mySum + sdata[tid + 32];
        sdata[tid] = mySum = mySum + sdata[tid + 16];
        sdata[tid] = mySum = mySum + sdata[tid +  8];
        sdata[tid] = mySum = mySum + sdata[tid +  4];
        sdata[tid] = mySum = mySum + sdata[tid +  2];
        sdata[tid] = mySum = mySum + sdata[tid +  1];
    }
    __syncthreads() ;
}

/// device function for products by parallel reduction in shared memory
/*!
  * Implementation based on NVIDIA whitepaper found at:
  * http://developer.download.nvidia.com/compute/cuda/1_1/Website/projects/reduction/doc/reduction.pdf
  *
  * Result is stored in sdata[0]
  \param sdata pointer to shared memory array
  \param my_factor factor loaded by the thread
  \param tid thread index
  */
__device__ void
productByReduction( volatile float* sdata, float my_factor, const unsigned int tid )
{
    sdata[tid] = my_factor;
    __syncthreads();

    // do reduction in shared mem
    if (tid < 128) { sdata[tid] = my_factor = my_factor * sdata[tid + 128]; } __syncthreads();
    if (tid <  64) { sdata[tid] = my_factor = my_factor * sdata[tid +  64]; } __syncthreads();

    if (tid < 32)
    {
        sdata[tid] = my_factor = my_factor * sdata[tid + 32];
        sdata[tid] = my_factor = my_factor * sdata[tid + 16];
        sdata[tid] = my_factor = my_factor * sdata[tid +  8];
        sdata[tid] = my_factor = my_factor * sdata[tid +  4];
        sdata[tid] = my_factor = my_factor * sdata[tid +  2];
        sdata[tid] = my_factor = my_factor * sdata[tid +  1];
    }
    __syncthreads() ;
}

/// device function for finding max value by parallel reduction in shared memory
/*!
  * Implementation based on NVIDIA whitepaper found at:
  * http://developer.download.nvidia.com/compute/cuda/1_1/Website/projects/reduction/doc/reduction.pdf
  *
  * Result is stored in sdata[0]. Other values in the array are garbage.
  \param sdata pointer to shared memory array
  \param val value loaded by the thread
  \param tid thread index
  */
__device__ void
maxByReduction( volatile float* sdata, float val, const unsigned int tid )
{
    sdata[tid] = val ;
    __syncthreads();

    // do reduction in shared mem
    if (tid < 128) { sdata[tid] = val = fmax(sdata[tid+128],val) ; } __syncthreads();
    if (tid <  64) { sdata[tid] = val = fmax(sdata[tid+64],val) ; } __syncthreads();

    if (tid < 32)
    {
        sdata[tid] = val = fmax(sdata[tid+32],val) ;
        sdata[tid] = val = fmax(sdata[tid+16],val) ;
        sdata[tid] = val = fmax(sdata[tid+8],val) ;
        sdata[tid] = val = fmax(sdata[tid+4],val) ;
        sdata[tid] = val = fmax(sdata[tid+2],val) ;
        sdata[tid] = val = fmax(sdata[tid+1],val) ;
    }
    __syncthreads() ;
}

__device__ float
logsumexpByReduction( volatile float* sdata, float val, const unsigned int tid )
{
    maxByReduction( sdata, val, tid ) ;
    float maxval = sdata[0] ;
    __syncthreads() ;

    sumByReduction( sdata, exp(val-maxval), tid) ;
    return safeLog(sdata[0]) + maxval ;
}


__device__ __host__
int sub_to_idx(int row, int col, int dim)
{
    int idx = row + col*dim ;
    return idx ;
}

template<class GaussianType>
__device__ __host__
void copy_gaussians(GaussianType &src, GaussianType &dest)
{
    // determine the size of the covariance matrix
    int dims = getGaussianDim(src) ;
    // copy mean and covariance
    for (int i = 0 ; i < dims*dims ; i++ )
    {
        if ( i < dims )
            dest.mean[i] = src.mean[i] ;
        dest.cov[i] = src.cov[i] ;
    }

    // copy weight
    dest.weight = src.weight ;
}

template<class GaussianType>
__device__ __host__
void force_symmetric_covariance(GaussianType &g)
{
    int dims = getGaussianDim(g) ;
    for ( int i = 0 ; i < dims ; i++ )
    {
        for( int j = 0 ; j < i ; j++)
        {
            int idx_lower = sub_to_idx(i,j,dims) ;
            int idx_upper = sub_to_idx(j,i,dims) ;
            g.cov[idx_lower] = (g.cov[idx_lower] + g.cov[idx_upper])/2 ;
            g.cov[idx_upper] = g.cov[idx_lower] ;
        }
    }
}

// explicit template instantiation
template __device__ __host__
void force_symmetric_covariance(Gaussian6D &g) ;

template __device__ __host__ void
clearGaussian(Gaussian6D &g) ;

template __device__ float
computeMahalDist(Gaussian6D a, Gaussian6D b) ;

template __device__ __host__ void
copy_gaussians(Gaussian6D &src, Gaussian6D &dest) ;

